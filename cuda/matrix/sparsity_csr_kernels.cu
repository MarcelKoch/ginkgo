#include "hip/hip_runtime.h"
/*******************************<GINKGO LICENSE>******************************
Copyright (c) 2017-2022, the Ginkgo authors
All rights reserved.

Redistribution and use in source and binary forms, with or without
modification, are permitted provided that the following conditions
are met:

1. Redistributions of source code must retain the above copyright
notice, this list of conditions and the following disclaimer.

2. Redistributions in binary form must reproduce the above copyright
notice, this list of conditions and the following disclaimer in the
documentation and/or other materials provided with the distribution.

3. Neither the name of the copyright holder nor the names of its
contributors may be used to endorse or promote products derived from
this software without specific prior written permission.

THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS
IS" AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED
TO, THE IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A
PARTICULAR PURPOSE ARE DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT
HOLDER OR CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL,
SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT
LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE,
DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY
THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
(INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
******************************<GINKGO LICENSE>*******************************/

#include "core/matrix/sparsity_csr_kernels.hpp"


#include <ginkgo/core/base/exception_helpers.hpp>


#include "accessor/reduced_row_major.hpp"
#include "core/base/mixed_precision_types.hpp"
#include "core/synthesizer/implementation_selection.hpp"
#include "cuda/base/config.hpp"
#include "cuda/base/math.hpp"
#include "cuda/base/types.hpp"
#include "cuda/components/cooperative_groups.cuh"
#include "cuda/components/reduction.cuh"
#include "cuda/components/thread_ids.cuh"
#include "cuda/components/uninitialized_array.hpp"


namespace gko {
namespace kernels {
namespace cuda {
/**
 * @brief The Compressed sparse row matrix format namespace.
 *
 * @ingroup sparsity
 */
namespace sparsity_csr {


constexpr int classical_overweight = 32;
constexpr int spmv_block_size = 128;
constexpr int warps_in_block = 4;


using classical_kernels = syn::value_list<int, 2>;


#include "common/cuda_hip/matrix/sparsity_csr_kernels.hpp.inc"


template <int dim, typename Type1, typename Type2>
GKO_INLINE auto as_cuda_accessor(
    const acc::range<acc::reduced_row_major<dim, Type1, Type2>>& acc)
{
    return acc::range<
        acc::reduced_row_major<dim, cuda_type<Type1>, cuda_type<Type2>>>(
        acc.get_accessor().get_size(),
        as_cuda_type(acc.get_accessor().get_stored_data()),
        acc.get_accessor().get_stride());
}


namespace host_kernel {


template <int subwarp_size, typename MatrixValueType, typename InputValueType,
          typename OutputValueType, typename IndexType>
void classical_spmv(syn::value_list<int, subwarp_size>,
                    std::shared_ptr<const CudaExecutor> exec,
                    const matrix::SparsityCsr<MatrixValueType, IndexType>* a,
                    const matrix::Dense<InputValueType>* b,
                    matrix::Dense<OutputValueType>* c,
                    const matrix::Dense<MatrixValueType>* alpha = nullptr,
                    const matrix::Dense<OutputValueType>* beta = nullptr)
{
    using input_accessor =
        gko::acc::reduced_row_major<2, OutputValueType, const InputValueType>;

    const auto nwarps = exec->get_num_warps_per_sm() *
                        exec->get_num_multiprocessor() * classical_overweight;
    const auto gridx =
        std::min(ceildiv(a->get_size()[0], spmv_block_size / subwarp_size),
                 int64(nwarps / warps_in_block));
    const dim3 grid(gridx, b->get_size()[1]);
    const dim3 block(spmv_block_size);

    const auto b_vals = gko::acc::range<input_accessor>(
        std::array<size_type, 2>{{b->get_size()[0], b->get_size()[1]}},
        b->get_const_values(), std::array<size_type, 1>{{b->get_stride()}});

    if (alpha == nullptr && beta == nullptr) {
        kernel::abstract_classical_spmv<subwarp_size><<<grid, block, 0, 0>>>(
            a->get_size()[0], as_cuda_type(a->get_const_value()),
            a->get_const_col_idxs(), as_cuda_type(a->get_const_row_ptrs()),
            as_cuda_accessor(b_vals), as_cuda_type(c->get_values()),
            c->get_stride());

    } else if (alpha != nullptr && beta != nullptr) {
        kernel::abstract_classical_spmv<subwarp_size><<<grid, block, 0, 0>>>(
            a->get_size()[0], as_cuda_type(alpha->get_const_values()),
            as_cuda_type(a->get_const_value()), a->get_const_col_idxs(),
            as_cuda_type(a->get_const_row_ptrs()), as_cuda_accessor(b_vals),
            as_cuda_type(beta->get_const_values()),
            as_cuda_type(c->get_values()), c->get_stride());
    } else {
        GKO_KERNEL_NOT_FOUND;
    }
}

GKO_ENABLE_IMPLEMENTATION_SELECTION(select_classical_spmv, classical_spmv);


}  // namespace host_kernel

template <typename MatrixValueType, typename InputValueType,
          typename OutputValueType, typename IndexType>
void spmv(std::shared_ptr<const CudaExecutor> exec,
          const matrix::SparsityCsr<MatrixValueType, IndexType>* a,
          const matrix::Dense<InputValueType>* b,
          matrix::Dense<OutputValueType>* c)
{
    host_kernel::select_classical_spmv(
        classical_kernels(), [](int compiled_info) { return true; },
        syn::value_list<int>(), syn::type_list<>(), exec, a, b, c);
}

GKO_INSTANTIATE_FOR_EACH_MIXED_VALUE_AND_INDEX_TYPE(
    GKO_DECLARE_SPARSITY_CSR_SPMV_KERNEL);


template <typename MatrixValueType, typename InputValueType,
          typename OutputValueType, typename IndexType>
void advanced_spmv(std::shared_ptr<const CudaExecutor> exec,
                   const matrix::Dense<MatrixValueType>* alpha,
                   const matrix::SparsityCsr<MatrixValueType, IndexType>* a,
                   const matrix::Dense<InputValueType>* b,
                   const matrix::Dense<OutputValueType>* beta,
                   matrix::Dense<OutputValueType>* c)
{
    host_kernel::select_classical_spmv(
        classical_kernels(), [](int compiled_info) { return true; },
        syn::value_list<int>(), syn::type_list<>(), exec, a, b, c, alpha, beta);
}

GKO_INSTANTIATE_FOR_EACH_MIXED_VALUE_AND_INDEX_TYPE(
    GKO_DECLARE_SPARSITY_CSR_ADVANCED_SPMV_KERNEL);


template <typename ValueType, typename IndexType>
void fill_in_dense(std::shared_ptr<const DefaultExecutor> exec,
                   const matrix::SparsityCsr<ValueType, IndexType>* input,
                   matrix::Dense<ValueType>* output) GKO_NOT_IMPLEMENTED;

GKO_INSTANTIATE_FOR_EACH_VALUE_AND_INDEX_TYPE(
    GKO_DECLARE_SPARSITY_CSR_FILL_IN_DENSE_KERNEL);


template <typename ValueType, typename IndexType>
void count_num_diagonal_elements(
    std::shared_ptr<const CudaExecutor> exec,
    const matrix::SparsityCsr<ValueType, IndexType>* matrix,
    size_type* num_diagonal_elements) GKO_NOT_IMPLEMENTED;

GKO_INSTANTIATE_FOR_EACH_VALUE_AND_INDEX_TYPE(
    GKO_DECLARE_SPARSITY_CSR_COUNT_NUM_DIAGONAL_ELEMENTS_KERNEL);


template <typename ValueType, typename IndexType>
void remove_diagonal_elements(
    std::shared_ptr<const CudaExecutor> exec, const IndexType* row_ptrs,
    const IndexType* col_idxs,
    matrix::SparsityCsr<ValueType, IndexType>* matrix) GKO_NOT_IMPLEMENTED;

GKO_INSTANTIATE_FOR_EACH_VALUE_AND_INDEX_TYPE(
    GKO_DECLARE_SPARSITY_CSR_REMOVE_DIAGONAL_ELEMENTS_KERNEL);


template <typename ValueType, typename IndexType>
void transpose(std::shared_ptr<const CudaExecutor> exec,
               const matrix::SparsityCsr<ValueType, IndexType>* orig,
               matrix::SparsityCsr<ValueType, IndexType>* trans)
    GKO_NOT_IMPLEMENTED;

GKO_INSTANTIATE_FOR_EACH_VALUE_AND_INDEX_TYPE(
    GKO_DECLARE_SPARSITY_CSR_TRANSPOSE_KERNEL);


template <typename ValueType, typename IndexType>
void sort_by_column_index(std::shared_ptr<const CudaExecutor> exec,
                          matrix::SparsityCsr<ValueType, IndexType>* to_sort)
    GKO_NOT_IMPLEMENTED;

GKO_INSTANTIATE_FOR_EACH_VALUE_AND_INDEX_TYPE(
    GKO_DECLARE_SPARSITY_CSR_SORT_BY_COLUMN_INDEX);


template <typename ValueType, typename IndexType>
void is_sorted_by_column_index(
    std::shared_ptr<const CudaExecutor> exec,
    const matrix::SparsityCsr<ValueType, IndexType>* to_check,
    bool* is_sorted) GKO_NOT_IMPLEMENTED;

GKO_INSTANTIATE_FOR_EACH_VALUE_AND_INDEX_TYPE(
    GKO_DECLARE_SPARSITY_CSR_IS_SORTED_BY_COLUMN_INDEX);


}  // namespace sparsity_csr
}  // namespace cuda
}  // namespace kernels
}  // namespace gko

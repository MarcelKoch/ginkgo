#include "hip/hip_runtime.h"
// SPDX-FileCopyrightText: 2017 - 2024 The Ginkgo authors
//
// SPDX-License-Identifier: BSD-3-Clause

#include "core/solver/batch_cg_kernels.hpp"


#include <thrust/functional.h>
#include <thrust/transform.h>


#include <ginkgo/core/base/exception_helpers.hpp>
#include <ginkgo/core/base/math.hpp>


#include "core/base/batch_struct.hpp"
#include "core/matrix/batch_struct.hpp"
#include "core/solver/batch_dispatch.hpp"
#include "cuda/base/batch_struct.hpp"
#include "cuda/base/config.hpp"
#include "cuda/base/kernel_config.hpp"
#include "cuda/base/thrust.cuh"
#include "cuda/base/types.hpp"
#include "cuda/components/cooperative_groups.cuh"
#include "cuda/components/reduction.cuh"
#include "cuda/components/thread_ids.cuh"
#include "cuda/components/uninitialized_array.hpp"
#include "cuda/matrix/batch_struct.hpp"


namespace gko {
namespace kernels {
namespace cuda {


// NOTE: this default block size is not used for the main solver kernel.
constexpr int default_block_size = 256;
constexpr int sm_oversubscription = 4;


/**
 * @brief The batch Cg solver namespace.
 *
 * @ingroup batch_cg
 */
namespace batch_cg {


#include "common/cuda_hip/base/batch_multi_vector_kernels.hpp.inc"
#include "common/cuda_hip/components/uninitialized_array.hpp.inc"
#include "common/cuda_hip/matrix/batch_csr_kernels.hpp.inc"
#include "common/cuda_hip/matrix/batch_dense_kernels.hpp.inc"
#include "common/cuda_hip/matrix/batch_ell_kernels.hpp.inc"


template <typename T>
using settings = gko::kernels::batch_cg::settings<T>;


template <typename ValueType>
void apply(std::shared_ptr<const DefaultExecutor> exec,
           const settings<remove_complex<ValueType>>& settings,
           const batch::BatchLinOp* const mat,
           const batch::BatchLinOp* const precon,
           const batch::MultiVector<ValueType>* const b,
           batch::MultiVector<ValueType>* const x,
           batch::log::detail::log_data<remove_complex<ValueType>>& logdata)
{
    GKO_NOT_IMPLEMENTED;
}

GKO_INSTANTIATE_FOR_EACH_VALUE_TYPE(GKO_DECLARE_BATCH_CG_APPLY_KERNEL);


}  // namespace batch_cg
}  // namespace cuda
}  // namespace kernels
}  // namespace gko
